#include "hip/hip_runtime.h"

#include <fstream>
#include "stdlib.h"
#include <stdio.h>


#define CSC(call)												\
do {															\
	hipError_t res = call;										\
	if (res != hipSuccess) {									\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",	    \
				__FILE__, __LINE__, hipGetErrorString(res));	\
		exit(0);												\
	}															\
} while (0)



__global__ void kernel(char* a, int n) {
	int idx = blockDim.x * blockIdx.x + threadIdx.x; //����� ������� ���� 
	printf("%s (current thread number %d)\n", a, idx);

}



int main() {
	const int n = 14;
	char chars[n] = "Hello, World!";

	char *arr_dev;
	CSC(hipMalloc(&arr_dev, sizeof(char) * n));
	CSC(hipMemcpy(arr_dev, chars, sizeof(char) * n, hipMemcpyHostToDevice));
	CSC(hipGetLastError());

	kernel << < 2, 4 >> > (arr_dev, n); 
	hipDeviceSynchronize();
	CSC(hipGetLastError());

	CSC(hipFree(arr_dev));

	return 0;

}
